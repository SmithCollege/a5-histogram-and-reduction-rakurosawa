
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define NUM_THDS 5
// #define BLOCK_SIZE 1024

__global__ void lessThdDiv(float* arr, char opperation){

    for (int i = NUM_THDS; i > 0; i /= 2){
        __syncthreads();
        if(threadIdx.x <= i){
            int idx1 = threadIdx.x * 2;
            float val1 = arr[idx1];
            int idx2 = (threadIdx.x * 2) + 1; 
            float val2 = arr[idx2];

            // MAX
            if (opperation == 'x'){
                if (val1 > val2){
                    arr[threadIdx.x] = val1;
                }
                else {
                    arr[threadIdx.x] = val2;
                }
            }

            // MIN
            else if (opperation == 'n'){
                if (val1 < val2){
                    arr[threadIdx.x] = val1;
                }
                else {
                    arr[threadIdx.x] = val2;
                }
            }

            // SUM
            else if (opperation == 's'){
                // adjusting values to prevent calculation errors 
                arr[idx1] = 0;
                arr[idx2] = 0;
                float locSum = val1 + val2;
                arr[threadIdx.x] = locSum;
            }

            // PROUDUCT
            else if (opperation == 'p'){
                // adjusting values to prevent calculation errors 
                arr[idx1] = 1;
                arr[idx2] = 1;
                float locProd = val1 * val2;
                arr[threadIdx.x] = locProd;
            }
        }
    }
    __syncthreads();

    // edge case of the last two values that still need to be dealt with:

    // MAX
    if (opperation == 'x' && arr[0] < arr[1]){
        arr[0] = arr[1];
    }

    // MIN
    else if (opperation == 'n' && arr[0] > arr[1]){
        arr[0] = arr[1];
    }

    // SUM
    else if (opperation == 's'){
        arr[0] = arr[0] + arr[1];
    }

    // PRODUCT
    else if (opperation == 'p'){
        arr[0] = arr[0] * arr[1];
    }

    __syncthreads();
}

int main(){

    float *input;
    hipMallocManaged(&input, (NUM_THDS*2)*sizeof(float));

    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = i;
    }

    lessThdDiv<<<1, NUM_THDS>>>(input, 'p');
    hipDeviceSynchronize();
    // printf("max value in arr is: %f\n", input[0]);
    // printf("min value in arr is: %f\n", input[0]);
    // printf("sum of arr is: %f\n", input[0]);
    printf("product arr is: %f\n", input[0]);

    hipFree(input);
}