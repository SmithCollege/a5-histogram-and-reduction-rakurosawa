
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define NUM_THDS 5
// #define BLOCK_SIZE 1024

__global__ void lessThdDiv(float* arr, char opperation){

    for (int i = NUM_THDS; i > 0; i /= 2){
        __syncthreads();
        if(threadIdx.x <= i){
            int idx1 = threadIdx.x * 2;
            float val1 = arr[idx1];
            int idx2 = (threadIdx.x * 2) + 1; 
            float val2 = arr[idx2];

            // MAX
            if (opperation == 'x'){
                if (val1 > val2){
                    arr[threadIdx.x] = val1;
                }
                else {
                    arr[threadIdx.x] = val2;
                }
            }
        }
    }
    __syncthreads();

    // edge case of the last two values still need to be dealt with:

    // MAX
    if (opperation == 'x'){
        if (arr[0] < arr[1]){
            arr[0] = arr[1];
        }
    }

    __syncthreads();
}

int main(){

    float *input;
    hipMallocManaged(&input, (NUM_THDS*2)*sizeof(float));

    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = i;
        printf("%f, ", input[i]);
    }

    printf("\n");

    lessThdDiv<<<1, NUM_THDS>>>(input, 'x');
    hipDeviceSynchronize();

    for (int i = 0; i < (NUM_THDS*2); i++) {
        printf("%f, ", input[i]);
    }
    printf("\n");

    hipFree(input);
}