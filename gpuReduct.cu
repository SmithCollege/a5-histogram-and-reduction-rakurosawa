
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define NUM_THDS 2048
#define BLOCK_SIZE 1024

__global__ void gpuReduct(float* arr, char opperation){
    
    int globalIdx = threadIdx.x + (BLOCK_SIZE * blockIdx.x);

    for (int i = 1; i < (NUM_THDS*2); i *= 2){
        __syncthreads();
        if (globalIdx % i == 0){
            // initialize values for threads to use
            int idx1 = globalIdx * 2 ;
            float val1 = arr[idx1];
            int idx2 = (globalIdx * 2) + i;
            float val2 = arr[idx2];

            // MAX
            if (opperation == 'x'){
                if (val1 > val2){
                    arr[idx1] = val1;
                }
                else {
                    arr[idx1] = val2;
                }
            }
            
            // MIN
            else if (opperation == 'n'){
                if (val1 < val2){
                    arr[idx1] = val1;
                }
                else {
                    arr[idx1] = val2;
                }
            }

            // SUM
            else if (opperation == 's'){
                float locSum = val1 + val2;
                arr[idx1] = locSum;
            }

            // PROUDUCT
            else if (opperation == 'p'){
                float locProd = val1 * val2;
                arr[idx1] = locProd;
            }
        }
    }
    
    __syncthreads();
        
}


int main(){

    float *input;
    hipMallocManaged(&input, (NUM_THDS*2)*sizeof(float));


    // MAX
    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = 1.0;
    }
    input[0] = 0.0;

    gpuReduct<<<NUM_THDS/BLOCK_SIZE, BLOCK_SIZE>>>(input, 'x');
    hipDeviceSynchronize();
    printf("max value in arr is: %f\n", input[0]);

    // MIN
    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = 1.0;
    }
    input[0] = 0.0;

    gpuReduct<<<NUM_THDS/BLOCK_SIZE, BLOCK_SIZE>>>(input, 'n');
    hipDeviceSynchronize();
    printf("min value in arr is: %f\n", input[0]);

    // SUM
    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = 1.0;
    }
    input[0] = 0.0;

    gpuReduct<<<NUM_THDS/BLOCK_SIZE, BLOCK_SIZE>>>(input, 's');
    hipDeviceSynchronize();
    printf("sum of arr is: %f\n", input[0]);

    // PRODUCT
    for (int i = 0; i < (NUM_THDS*2); i++) {
        input[i] = 1.0;
    }
    input[0] = 0.0;

    gpuReduct<<<NUM_THDS/BLOCK_SIZE, BLOCK_SIZE>>>(input, 'p');
    hipDeviceSynchronize();
    printf("product arr is: %f\n", input[0]);

    hipFree(input);
}